#include "hip/hip_runtime.h"
#include "support.h"

#define FILTER_WIDTH 3
__constant__ short int dc_x_sobel[FILTER_WIDTH * FILTER_WIDTH];
__constant__ short int dc_y_sobel[FILTER_WIDTH * FILTER_WIDTH];

__global__ void convert_RGB_to_energy(uchar3* original_image, uint32_t* energy_image, int width, int height){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ uchar3 shared_image[];

    if(row < height && col < width){
        // Copy to shared memory
        int s_size = blockDim.x + 2;
        int s_row = threadIdx.y + 1;
        int s_col = threadIdx.x + 1;

        shared_image[s_row * s_size + s_col] = original_image[row * width + col];

        // Padding edge cells
        int left = max(0, col - 1);
		int right = min(col + blockDim.x, width - 1);
		int top = max(0, row - 1);
		int bottom = min(row + blockDim.y, height - 1);

        if (threadIdx.x == 0) {
            // left and right edge in s_row
            shared_image[s_row * s_size] = original_image[row * width + left];
			shared_image[s_row * s_size + blockDim.x + 1] = original_image[row * width + right];
            if (threadIdx.y == 0) {
                // 4 corners of padding
                shared_image[0] = original_image[top * width + left];
				shared_image[blockDim.x + 1] = original_image[top * width + right];
				shared_image[(blockDim.y + 1) * s_size] = original_image[bottom * width + left];
				shared_image[(blockDim.y + 1) * s_size + blockDim.x + 1] = original_image[bottom * width + right];
            }
        }
        if (threadIdx.y == 0) {
            // top and bottom edge in s_col
            shared_image[threadIdx.x + 1] = original_image[top * width + col];
			shared_image[(blockDim.y + 1) * s_size + threadIdx.x + 1] = original_image[bottom * width + col];
        }

        __syncthreads();

        // Compute energy
        int x = 0, y = 0;
        for(int r = 0; r < FILTER_WIDTH; r++){
            for(int c = 0; c < FILTER_WIDTH; c++){
                uchar3 pixel = shared_image[(threadIdx.y + r) * s_size + threadIdx.x + c];
                uint32_t gray_pixel = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;
                x += gray_pixel * dc_x_sobel[r * FILTER_WIDTH + c];
                y += gray_pixel * dc_y_sobel[r * FILTER_WIDTH + c];
            }
        }
        energy_image[row * width + col] = abs(x) + abs(y);
    }
}

__global__ void find_seam(uint32_t* energy_image, uint32_t* back_tracking, int width, int height, int row){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t temp = UINT32_MAX;
    if(col < width){
        int p = row * width + col;
        for(int i = -1; i < 2; i++){
            int upper_col = min(max(col + i, 0), width - 1);
            if(energy_image[(row - 1) * width + upper_col] < temp){
                temp = energy_image[(row - 1) * width + upper_col];
                back_tracking[p] = upper_col;
            }
        }
        energy_image[p] += temp;
    }
}

__global__ void find_min_index(uint32_t* last_row, int n, uint32_t* min_indices){
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    extern __shared__ uint32_t s_last_row[];
    extern __shared__ uint32_t s_min_indices[];

    if (i < n) {
        s_last_row[i] = last_row[i];
        s_min_indices[threadIdx.x] = i;
    }
    else return;
    if (i + blockDim.x < n) {
        s_last_row[i + blockDim.x] = last_row[i + blockDim.x];
        s_min_indices[threadIdx.x + blockDim.x] = i + blockDim.x;
    }
    __syncthreads();

    // min reduce
    for (int stride = blockDim.x; stride > threadIdx.x; stride >>= 1) {
        if (i + stride < n) {
            if (last_row[s_min_indices[threadIdx.x]] > last_row[s_min_indices[threadIdx.x + stride]]) {
                s_min_indices[threadIdx.x] = s_min_indices[threadIdx.x + stride];
            }
        }
        
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        min_indices[blockIdx.x] = s_min_indices[0];
    }
}

__global__ void remove_seam(uchar3* in_image, uchar3* out_image, int width, int height, uint32_t* seam){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if(row < height && col < width - 1){
        if(col < seam[row])
            out_image[(width - 1) * row + col] = in_image[row * width + col];
        else
            out_image[(width - 1) * row + col] = in_image[row * width + col + 1];
    }
}
void remove_n_seam(uchar3* original_image, uchar3* out_image, int width, int height, int n_seams){
    short int x_sobel[FILTER_WIDTH * FILTER_WIDTH] = { 1,  0, -1,
					                                   2,  0, -2,
					                                   1,  0, -1};
	short int y_sobel[FILTER_WIDTH * FILTER_WIDTH] = { 1,  2,  1,
                                                       0,  0,  0,
                                                      -1, -2, -1};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_x_sobel), x_sobel, FILTER_WIDTH * FILTER_WIDTH * sizeof(short int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_y_sobel), y_sobel, FILTER_WIDTH * FILTER_WIDTH * sizeof(short int)));
    uchar3 *d_original_image;
    uint32_t *d_energy_image;
    uint32_t *d_back_tracking;
    uint32_t *d_seam;
    uchar3 * d_output_image;
    uint32_t *d_min_indices;
    size_t n_bytes_uchar3 = width * height * sizeof(uchar3);
    size_t n_bytes_uint32t = width * height * sizeof(uint32_t);
    // size_t n_bytes_row = width * sizeof(uint32_t);
    size_t n_bytes_height = height * sizeof(uint32_t);
    CHECK(hipMalloc(&d_original_image, n_bytes_uchar3));
    CHECK(hipMalloc(&d_energy_image, n_bytes_uint32t));
    CHECK(hipMalloc(&d_back_tracking, n_bytes_uint32t));
    CHECK(hipMalloc(&d_seam, n_bytes_height));
    CHECK(hipMalloc(&d_output_image, n_bytes_uchar3));
    CHECK(hipMemcpy(d_original_image, original_image, n_bytes_uchar3, hipMemcpyHostToDevice));
    uint32_t* seam = (uint32_t*)malloc(n_bytes_height);
    uint32_t* back_tracking = (uint32_t*)malloc(n_bytes_uint32t);
    dim3 block_size2d(32,32);
    dim3 grid_size2d((width - 1) / block_size2d.x + 1, (height - 1) / block_size2d.y + 1);
    dim3 block_size1d(512);
    dim3 grid_size1d((width - 1) / block_size1d.x + 1);
    uint32_t col_start_seam;
    
    for(int i = 0; i < n_seams; i++){
        // Convert RGB to gray and calculate energy
        convert_RGB_to_energy<<<grid_size2d, block_size2d, (block_size2d.x + 2) * (block_size2d.y + 2) * sizeof(uchar3)>>>(d_original_image, d_energy_image, width, height);
        
        // Find all seam (from top row to bottom row)
        for(int row = 1; row < height; row++)
            find_seam<<<grid_size1d, block_size1d>>>(d_energy_image, d_back_tracking, width, height, row);

        // Find min seam
        dim3 blockSize(128);
        dim3 gridSize((width - 1) / (2 * blockSize.x) + 1);
        CHECK(hipMalloc(&d_min_indices, gridSize.x * sizeof(uint32_t)));
        uint32_t *min_indices = (uint32_t*)malloc(gridSize.x * sizeof(uint32_t));
        uint32_t *last_row = d_energy_image + width * (height - 1);

        find_min_index<<<gridSize, block_size1d, (2 * blockSize.x + width) * sizeof(uint32_t)>>>(last_row, width, d_min_indices);
        CHECK(hipMemcpy(min_indices, d_min_indices, gridSize.x * sizeof(uint32_t), hipMemcpyDeviceToHost));

        col_start_seam = min_indices[0];
        last_row = (uint32_t*)malloc(width * sizeof(uint32_t));
        CHECK(hipMemcpy(last_row, d_energy_image + width * (height - 1), width * sizeof(uint32_t), hipMemcpyDeviceToHost));
        for (int j = 1; j < gridSize.x; ++j) {
            if (last_row[min_indices[j]] < last_row[col_start_seam])
                col_start_seam = min_indices[j];
        }
        hipDeviceSynchronize();
		CHECK(hipGetLastError());
        CHECK(hipFree(d_min_indices));
        free(min_indices);
        free(last_row);
        // Get seam to delete from backtracking
        CHECK(hipMemcpy(back_tracking, d_back_tracking, n_bytes_uint32t, hipMemcpyDeviceToHost));
        seam[height - 1] = col_start_seam;
        for(int row = height - 1; row > 0; row--)
            seam[row - 1] = back_tracking[row * width + seam[row]];

        // Remove seam from image
        CHECK(hipMemcpy(d_seam, seam, n_bytes_height, hipMemcpyHostToDevice));
        remove_seam<<<grid_size2d, block_size2d>>>(d_original_image, d_output_image, width, height, d_seam);
        uchar3* temp = d_original_image;
        d_original_image = d_output_image;
        d_output_image = temp;
        width -= 1;
    }
    
    CHECK(hipMemcpy(out_image, d_original_image, sizeof(uchar3) * width * height, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_original_image));
    CHECK(hipFree(d_energy_image));
    CHECK(hipFree(d_back_tracking));
    CHECK(hipFree(d_seam));
    CHECK(hipFree(d_output_image));
    free(seam);
    free(back_tracking);
}

int main(int argc, char** argv){
    printDeviceInfo();
	int width, height;
	uchar3* original_image;
	readPnm(argv[1], width, height, original_image);
	char* file_name_out  = strtok(argv[1], ".");
	// int n_seams = 5;
	int n_seams = argc == 3 ? atoi(argv[2]) : 100;
	printf("Image size (width x height): %i x %i\n\n", width, height);
    uchar3 *output_image = (uchar3*)malloc((width - n_seams) * height * sizeof(uchar3));

    GpuTimer timer;
    timer.Start();
    remove_n_seam(original_image, output_image, width, height, n_seams);
	timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
	printf("Output image size (width x height): %i x %i\n\n", width - n_seams, height);

    writePnm(output_image, width - n_seams, height, concatStr(file_name_out,"_device6.pnm"));

    free(original_image);
    free(output_image);
    return EXIT_SUCCESS;
}